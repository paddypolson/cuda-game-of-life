#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <iostream>
#include <tuple>
#include <random>
#include <functional>

__device__ int getNeighbourCount( bool** input, int x, int y, int* size ) {
    int count = 0;

    if ( ( x - 1 ) < 0 ) {
        if ( input[y][ size[1] ] ) { count++; }
    } else {
        if ( input[y][x - 1] ) { count++; }
    }
    if ( ( x + 1 ) >= size[1] ) {
        if ( input[y][0] ) { count++; }
    } else {
        if ( input[y][x + 1] ) { count++; }
    }
    if ( ( y - 1 ) < 0 ) {
        if ( input[ size[0] ][x] ) { count++; }
    } else {
        if ( input[y - 1][x] ) { count++; }
    }
    if ( ( x + 1 ) >= size[1] ) {
        if ( input[0][x] ) { count++; }
    } else {
        if ( input[y + 1][x] ) { count++; }
    }
    return count;
}

__global__ void simulate( bool* input, bool** output, int* size, int steps ) {

    int index = threadIdx.x;
    int stride = blockDim.x;

}

/*
Clears screen and moves cursor to home pos on POSIX systems
*/
void clear() {
    std::cout << "\033[2J;" << "\033[1;1H";
}

/*
*/
void printGrid( bool** grid, int* size ) {
    for ( int y = 0; y < size[1]; y++ ) {
        for ( int x = 0; x < size[0]; x++ ) {
            if ( grid[y][x] == true ) {
                std::cout << "0";
            }
            else {
                std::cout << ".";
            }
        }
        std::cout << std::endl;
    }
}

static void show_usage(std::string name)
{
    std::cerr << "Usage: " << name << " [-i input.txt]/[-r] [-o output.txt] [-s 10]\n"
              << "Options:" << std::endl
              << "\t-h, --help\t\tShow this help message and exit" << std::endl
              << "\t-i, --input\t\tProvide an input file for the starting state" << std::endl
              << "\t-r, --random\t\tInstead start with a randomized starting state, provide a seed, 0 will set a random seed" << std::endl
              << "\t-o, --output\t\tOptionally save the final state as a file" << std::endl
              << "\t-s, --steps\t\tThe number of simulation step to take" << std::endl
              << std::endl;
}

int main( int argc, char* argv[] ) {

    int opt;
    char* input;
    char* output;
    bool isRandom = false;
    int seed;
    int steps;
    int size[2] = {10, 10};

    bool** grid;
    bool** d_in;        // The read-only input array for kernel
    bool** d_out;       // The write-only output for kernel

    if ( argc < 2 ) {
        show_usage( argv[0] );
        exit( EXIT_FAILURE );
    }

    while (( opt = getopt(argc, argv, "hi:o:r:s:" )) != -1 ) {
        switch ( opt ) {

        case 'h':
            show_usage( argv[0] );
            exit( EXIT_FAILURE );
            break;

        case 'i':
            input = optarg;
            break;
        
        case 'o':
            output = optarg;
            break;

        case 'r':
            isRandom = true;
            seed = atoi(optarg);
            break;

        case 's':
            steps = atoi(optarg);
            break;

        default: /* '?' */
        show_usage( argv[0] );
        exit( EXIT_FAILURE );
        }
    }

    // Init empty grid
    grid = (bool**) malloc( size[1] * sizeof(bool*) );
    hipMalloc( &d_in, size[1] * size[0] * sizeof(bool*) );
    hipMalloc( &d_out, size[1] * size[0] * sizeof(bool*) );

    for ( int y = 0; y < size[1]; y++ ) {
        grid[y] = (bool*) malloc( size[0] * sizeof(bool) );

        for ( int x = 0; x < size[0]; x++ ) {
            grid[y][x] = false; // Init host grid to empty
        }
    }

    if ( isRandom ) {
        auto gen = std::bind(   std::uniform_int_distribution<>( 0,1 ),
                                std::default_random_engine() );
        for ( int y = 0; y < size[1]; y++ ) {    
            for ( int x = 0; x < size[0]; x++ ) {
                grid[y][x] = gen();
            }
        }
    }

    printGrid( grid, size );

    simulate<<<1,1>>>();
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Clean up memory allocations
    for ( int y = 0; y < size[1]; y++ ) {
        free( grid[y] );
    }
    free( grid );
    hipFree( d_in );
    hipFree( d_out );

    exit( EXIT_SUCCESS );
}
